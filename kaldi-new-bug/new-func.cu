
#include <hip/hip_runtime.h>
__global__
void _copy_low_upp(float* A, int rows, int stride) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i <= j || i >= rows)
    return;
  int index_1 = i * stride + j;
  int index_2 = j * stride + i;
  A[index_2] = A[index_1];
}
// rows = 5, stride = 0, block = (2, 1, 1), thread = (3, 2, 2)
// (0, 0, 0) (1, 0, 1) with (0, 0, 0) (1, 0, 0)
// i = 1, j = 0, index_1 = 0, index_2 = 1; i = 1, j = 0, index_1 = 0, index_2 = 1
// (2, 1, 0) read, (1, 0, 1) write. (1 0 1) write to 1, (2 1 0) read from 1 



__global__
void _copy_upp_low(float* A, int rows, int stride) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (j <= i || j >= rows)
    return;
  int index_1 = i * stride + j;
  int index_2 = j * stride + i;
  A[index_2] = A[index_1];
}


__global__
void _add_diag_vec_mat(float alpha, float *mat, int stride, int rows, int cols,
                              const float *vec, const float *mat2,
                              int mat2_row_stride, int mat2_col_stride,
                              float beta) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;  
  int j = blockIdx.y * blockDim.y + threadIdx.y;  

  int index = j * stride + i, index2 = j * mat2_row_stride
      + i * mat2_col_stride;

  if (i < cols && j < rows) {
    mat[index] = alpha * vec[j] * mat2[index2] + beta * mat[index];
  }
}